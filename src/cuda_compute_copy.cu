#include <stdint.h>
#include <hip/hip_runtime.h>

#define UNROLL 8
#define WARP_SIZE 32

__global__ static void simple_copy(void *dst_v, void *src_v, size_t size)
{
    ulong2 *dst = (ulong2 *)dst_v;
    ulong2 *src = (ulong2 *)src_v;

    int nwarps =  gridDim.x * blockDim.x / WARP_SIZE;
    int warp = (blockDim.x * blockIdx.x + threadIdx.x) / WARP_SIZE;
    int wid = threadIdx.x % WARP_SIZE;

    size /= sizeof(*src);

    ulong2 *s = src + warp * WARP_SIZE * UNROLL;
    ulong2 *d = dst + warp * WARP_SIZE * UNROLL;

    while (s < src + size) {
        // It's faster to do a bunch of reads, followed by a bunch of writes,
        // instead of going one by one.
        ulong2 data[UNROLL];

        #pragma unroll
        for (int u=0; u<UNROLL; u++) {
            data[u] = s[u * WARP_SIZE + wid];
        }

        #pragma unroll
        for (int u=0; u<UNROLL; u++) {
            d[u * WARP_SIZE + wid] = data[u];
        }

        s += nwarps * WARP_SIZE * UNROLL;
        d += nwarps * WARP_SIZE * UNROLL;
    }
}

extern "C" void ComputeCopy(hipStream_t stream, void *dst_v, void *src_v, size_t size)
{
    // Decent numbers for A100, haven't tried anything else.
    simple_copy<<<14, 512, 0, stream>>>(dst_v, src_v, size);
}
